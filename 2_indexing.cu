#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void sum(float *x)
{
    // 泛指当前block在所有block范围内的id
    int block_id = blockIdx.x;
    // 泛指当前线程在所有block范围内的全局id
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    // 泛指当前线程在其block内的id
    int local_tid = threadIdx.x;
    printf("current block=%d, thread id in current block =%d, global thread id=%d\n", block_id, local_tid, global_tid);
    x[global_tid] += 1;
}

int main(){
    int N = 32;
    int nbytes = N * sizeof(float);
    float *dx, *hx;
    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);//传二级指针是为了能修改dx中的值，在dx中存储分配空间的起始地址
    /* allocate CPU mem */
    hx = (float*) malloc(nbytes);
    /* init host data */
    printf("hx original: \n");
    for (int i = 0; i < N; i++) {
        hx[i] = i;
        cout << hx[i] << endl;
    }
    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    /* launch GPU kernel */
    sum<<<1, N>>>(dx);
    /* copy data from GPU */
    hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);
    printf("hx current: \n");
    for (int i = 0; i < N; i++) {
        cout << hx[i] << endl;
    }
    hipFree(dx);
    free(hx);
    return 0;
}
